#include "hip/hip_runtime.h"
// nvcc -o testing -lglut -lGL -lm testing.cu

#include <stdio.h>
#ifndef __GNUC__
	#include <GL/freeglut.h>
	typedef __int32 int32_t;
	typedef unsigned __int32 uint32_t;
	typedef unsigned __int16 uint16_t;
#else
	#include <stdint.h>
	#include <GL/glut.h>
#endif

// image sizes
#define IMG_WIDTH	400
#define IMG_HEIGHT	300
#ifndef M_PI
#define M_PI  3.14159265358979323846
#endif

// Hough transform parameters
#define HS_STEP		1
#define HS_ANGLES	(90/HS_STEP + 1)
#define HS_1_WIDTH	(IMG_HEIGHT + IMG_WIDTH + IMG_HEIGHT)
#define HS_1_SIZE	(HS_1_WIDTH * HS_ANGLES)
#define HS_2_WIDTH	(IMG_WIDTH + IMG_HEIGHT + IMG_WIDTH)
#define HS_2_SIZE	(HS_2_WIDTH * HS_ANGLES)

#define THREADS_X 	32
#define THREADS_Y	4
#define BLOCKS_X 	(IMG_WIDTH  / THREADS_X)
#define BLOCKS_Y 	(IMG_HEIGHT / THREADS_Y)

typedef struct {
	uint32_t filesz;
	uint16_t creator1;
	uint16_t creator2;
	uint32_t bmp_offset;
} bmp_file_header;

typedef struct {
	uint32_t header_sz;
	int32_t width;
	int32_t height;
	uint16_t nplanes;
	uint16_t bitspp;
	uint32_t compress_type;
	uint32_t bmp_bytesz;
	int32_t hres;
	int32_t vres;
	uint32_t ncolors;
	uint32_t nimpcolors;
} bmp_info_header;

unsigned char *input, *output1, *output2;
unsigned char *dev_input, *dev_output, *dev_grey, *dev_edges;
unsigned int *houghspace1, *houghspace2;
int n_max1, b_max1, n_max2, b_max2;

static void handle_error(hipError_t err) {
	if (err != 0) {
		fprintf(stderr, "%sparam\n", hipGetErrorString(err));
		exit(err);
	}
}

void allocate_memory() {
	int n_pixels = IMG_WIDTH * IMG_HEIGHT;
	int image_size = sizeof(char) * n_pixels * 3;
	input = (unsigned char*) malloc(image_size);
	output1 = (unsigned char*) malloc(image_size);
	output2 = (unsigned char*) malloc(image_size);
	houghspace1 = (unsigned int*) malloc(HS_1_SIZE*sizeof(unsigned int));
	houghspace2 = (unsigned int*) malloc(HS_2_SIZE*sizeof(unsigned int));
	handle_error(hipMalloc(&dev_input, image_size));
	handle_error(hipMalloc(&dev_output, image_size));
	handle_error(hipMalloc(&dev_grey, n_pixels));
	handle_error(hipMalloc(&dev_edges, n_pixels));
}

__global__ void colour_threshold(unsigned char *bgr, unsigned char *greyscale, int n_pixels, unsigned char b, unsigned char g, unsigned char r, int threshold) {

	int thread_id = (blockIdx.x * blockDim.x) + (threadIdx.x);

	if (thread_id < n_pixels) {
		unsigned char *pixel = &bgr[thread_id * 3];
		int db = *pixel++ -b;
		int dg = *pixel++ -g;
		int dr = *pixel   -r;
		int distance = (db*db) + (dg*dg) + (dr*dr);

		if (distance <= threshold) {
			greyscale[thread_id] = 255;
		} else {
			greyscale[thread_id] = 0;
		}
	}
}

__global__ void greyscale_to_bgr(unsigned char *greyscale, unsigned char *bgr, int n_pixels) {

	int thread_id = (blockIdx.x * blockDim.x) + (threadIdx.x);

	if (thread_id < n_pixels) {
		bgr = &bgr[thread_id * 3];
		bgr[0] = greyscale[thread_id];
		bgr[1] = greyscale[thread_id];
		bgr[2] = greyscale[thread_id];
	}
}

__global__ void detect_edges(unsigned char *in, unsigned char *out, int n_pixels, int width, int height) {
	int thread_id = (blockIdx.x * blockDim.x) + (threadIdx.x);

	if (thread_id < n_pixels) {
		int b, d, f, h, r, x, y;

		y = thread_id / width;
		x = thread_id - (width * y);

		if (x==0 || y==0 || x==width-1 || y==height-1) {
			out[thread_id] = 0;
		} else {
			b = thread_id + width;
			d = thread_id - 1;
			f = thread_id + 1;
			h = thread_id - width;

			r = 0;

			if (in[thread_id]) {
				r+=4;
			}

			if (in[b]) {
				r=r-1;
			}
			if (in[d]) {
				r=r-1;
			}
			if (in[f]) {
				r=r-1;
			}
			if (in[h]) {
				r=r-1;
			}

			if (r>0) {
				out[thread_id]=255;
			} else {
				out[thread_id]=0;
			}
		}
	}
}

__global__ void kHough4(unsigned char const * const image, unsigned int* const houghspace1, unsigned int* const houghspace2) {
	int const x = blockIdx.x * blockDim.x + threadIdx.x;
	int const y = blockIdx.y * blockDim.y + threadIdx.y;
	
	//cache all possible values of M
	__shared__ float sh_m_array[THREADS_X*THREADS_Y];
	int const n = threadIdx.y*THREADS_X + threadIdx.x;
	sh_m_array[n] = (n-((HS_ANGLES-1)/2.0f)) / (float)((HS_ANGLES-1)/2.0f);
	__syncthreads();
	
	//read one image pixel from global memory
	unsigned char pixel = image[y*IMG_WIDTH + x];
	
	//vote for each non zero pixel
	if (pixel > 0)
	{
		for (int n = 0; n < HS_ANGLES; n++)
		{
			float const m = sh_m_array[n];
			int const b1 = x - (int)(y*m) + IMG_HEIGHT;
			int const b2 = y - (int)(x*m) + IMG_WIDTH;

			houghspace1[n*HS_1_WIDTH+b1]++;
			houghspace2[n*HS_2_WIDTH+b2]++;
		}
	}
}

void calcHough(unsigned int* houghspace1, unsigned int* houghspace2) {
	unsigned int *ghoughspace1, *ghoughspace2;

	//set cache configuration
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kHough4), hipFuncCachePreferShared);
	
	//allocate variables on device (GPU)
	handle_error(hipMalloc((void**)&ghoughspace1, HS_1_SIZE*sizeof(unsigned int)));
	handle_error(hipMalloc((void**)&ghoughspace2, HS_2_SIZE*sizeof(unsigned int)));

	//reset Hough space
	handle_error(hipMemset(ghoughspace1, 0, HS_1_SIZE*sizeof(unsigned int)));
	handle_error(hipMemset(ghoughspace2, 0, HS_2_SIZE*sizeof(unsigned int)));

	//run kernel
	dim3 dimBlock1(THREADS_X, THREADS_Y);
	dim3 dimGrid1(BLOCKS_X, BLOCKS_Y);
	kHough4<<<dimGrid1, dimBlock1>>>(dev_edges, ghoughspace1, ghoughspace2);

	//copy the GPU results back to CPU
	handle_error(hipMemcpy(houghspace1, ghoughspace1, HS_1_SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost));
	handle_error(hipMemcpy(houghspace2, ghoughspace2, HS_2_SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost));

	//free GPU memory
	handle_error(hipFree(ghoughspace1));
	handle_error(hipFree(ghoughspace2));
}

void calculate_output() {
	int n_pixels = IMG_WIDTH * IMG_HEIGHT;
	int image_size = sizeof(char) * n_pixels * 3;

	handle_error(hipMemcpy(dev_input, input, image_size, hipMemcpyHostToDevice));

	colour_threshold<<<IMG_WIDTH, IMG_HEIGHT>>>(dev_input, dev_grey, n_pixels, 0, 255, 255, 20000);
	greyscale_to_bgr<<<IMG_WIDTH, IMG_HEIGHT>>>(dev_grey, dev_output, n_pixels);
	handle_error(hipMemcpy(output1, dev_output, image_size, hipMemcpyDeviceToHost));

	detect_edges<<<IMG_WIDTH, IMG_HEIGHT>>>(dev_grey, dev_edges, n_pixels, IMG_WIDTH, IMG_HEIGHT);
	greyscale_to_bgr<<<IMG_WIDTH, IMG_HEIGHT>>>(dev_edges, dev_output, n_pixels);
	handle_error(hipMemcpy(output2, dev_output, image_size, hipMemcpyDeviceToHost));

	calcHough(houghspace1, houghspace2);
	unsigned int value, max_hs1=0, max_hs2=0;
	int n, b;
	for (n = 0; n < HS_ANGLES; n++) //found the maximum value in Hough space
	{
		for (b = 0; b < HS_1_WIDTH; b++)
		{
			value = houghspace1[n*HS_1_WIDTH+b];
			if (value > max_hs1)
			{
				max_hs1 = value;
				n_max1 = n; b_max1 = b;
			}
		}
		for (b = 0; b < HS_2_WIDTH; b++)
		{
			value = houghspace2[n*HS_2_WIDTH+b];
			if (value > max_hs2)
			{
				max_hs2 = value;
				n_max2 = n; b_max2 = b;
			}
		}
	}
}

void tidy_and_exit() {
	free(input);
	free(output1);
	free(output2);
	free(houghspace1);
	free(houghspace2);
	handle_error(hipFree(dev_input));
	handle_error(hipFree(dev_output));
	handle_error(hipFree(dev_grey));
	handle_error(hipFree(dev_edges));
	exit(0);
}

#define YSHIFT 95
static void display() {
	unsigned char *temp;

	glClear(GL_COLOR_BUFFER_BIT);

	glRasterPos2i(0, YSHIFT);
	glDrawPixels(IMG_WIDTH, IMG_HEIGHT, GL_BGR_EXT, GL_UNSIGNED_BYTE, input);

	glRasterPos2i(IMG_WIDTH, YSHIFT);
	glDrawPixels(IMG_WIDTH, IMG_HEIGHT, GL_BGR_EXT, GL_UNSIGNED_BYTE, output2);

	glRasterPos2i(2*IMG_WIDTH, YSHIFT);
	glDrawPixels(IMG_WIDTH, IMG_HEIGHT, GL_BGR_EXT, GL_UNSIGNED_BYTE, output1);

	glRasterPos2i(3*IMG_WIDTH, YSHIFT);
	glDrawPixels(IMG_WIDTH, IMG_HEIGHT, GL_BGR_EXT, GL_UNSIGNED_BYTE, input);
	float tanth, b, x1, y1, x2, y2;
	x1 = 0.0; x2 = 1.0;
	tanth = tan((n_max1-45)*M_PI/180);
	b = (float)b_max1/(IMG_WIDTH+IMG_HEIGHT+2*HS_ANGLES);
	y1 = tanth*x1 + b; y2 = tanth*x2 + b;
	//scaling and shifted to raster coordinates
	x1 = x1*IMG_WIDTH + 3*IMG_WIDTH; x2 = x2*IMG_WIDTH + 3*IMG_WIDTH;
	y1 = y1*IMG_HEIGHT + YSHIFT; y2 = y2*IMG_HEIGHT + YSHIFT;
	glColor3ub(255, 0, 0); //red color
	glBegin(GL_LINES); //draw line
		glVertex2f(x1, y1);
		glVertex2f(x2, y2);
	glEnd();
	x1 = 0.0; x2 = 1.0;
	tanth = tan((n_max2-45)*M_PI/180);
	b = (float)IMG_WIDTH/(2*IMG_HEIGHT) - (float)b_max2/(IMG_WIDTH+IMG_HEIGHT+2*HS_ANGLES);
	y1 = tanth*x1 + b; y2 = tanth*x2 + b;
	x1 = x1*IMG_WIDTH + 3*IMG_WIDTH; x2 = x2*IMG_WIDTH + 3*IMG_WIDTH;
	y1 = y1*IMG_HEIGHT + YSHIFT; y2 = y2*IMG_HEIGHT + YSHIFT;
	glBegin(GL_LINES);
		glVertex2f(x1, y1);
		glVertex2f(x2, y2);
	glEnd();

	temp = (unsigned char*) malloc(HS_1_SIZE*sizeof(unsigned char));
	for (int i = 0; i < HS_1_SIZE; i++) temp[i] = 8*houghspace1[i];
	glRasterPos2i(0, 0);
	glDrawPixels(HS_1_WIDTH, HS_ANGLES, GL_LUMINANCE, GL_UNSIGNED_BYTE, temp);
	free(temp);

	temp = (unsigned char*) malloc(HS_2_SIZE*sizeof(unsigned char));
	for (int i = 0; i < HS_2_SIZE; i++) temp[i] = 8*houghspace2[i];
	glRasterPos2i(HS_1_WIDTH, 0);
	glDrawPixels(HS_2_WIDTH, HS_ANGLES, GL_LUMINANCE, GL_UNSIGNED_BYTE, temp);
	free(temp);

	glFlush();
}

void load_image_data(char* filename) {
	FILE *f = fopen(filename, "rb");
	if (!f) {
		printf("failed to open file\n");
		exit(0);
	}

	char signature[2];
	fread(signature, 1, 2, f);
	bmp_file_header bfh;
	fread(&bfh, 1, sizeof(bmp_file_header), f);
	bmp_info_header bih;
	fread(&bih, 1, sizeof(bmp_info_header), f);

	if (bih.width != IMG_WIDTH || bih.height != IMG_HEIGHT)
	{
		printf("Error: unexpected image size (%d x %d)\n", bih.width, bih.height);
		printf("       expected (%d x %d)\n", IMG_WIDTH, IMG_HEIGHT);
		fclose(f);
		exit(0);
	}

	allocate_memory();

	fread(input, 1, IMG_WIDTH*IMG_HEIGHT*3, f);

	fclose(f);
}

static void key_pressed(unsigned char key, int x, int y) {
	switch (key) {
	case 27: // escape
		tidy_and_exit();
		break;
	default:
		printf("\nPress escape to exit\n");
		break;
	}
}

int main(int argc, char **argv) {
	load_image_data("2pencils.bmp");
	calculate_output();

	glutInit(&argc, argv);
	glutInitWindowSize(IMG_WIDTH*4, IMG_HEIGHT+YSHIFT);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
	glutCreateWindow("Hough lines");
	glutDisplayFunc(display);
	glutKeyboardFunc(key_pressed);
	glOrtho(0, IMG_WIDTH*4, 0, IMG_HEIGHT, 0, 1);
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glutMainLoop();

	tidy_and_exit();

	return 0;
}
